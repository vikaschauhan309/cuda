
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloFromGPU() {
    printf("Hello from GPU thread %d!\n", threadIdx.x);
}

int main() {
    std::cout << "Hello from CPU!" << std::endl;

    helloFromGPU<<<1, 5>>>();
    hipDeviceSynchronize();

    return 0;
}
