#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "matrix_add.hpp"

__global__ void matrixAddKernel(const float* A, const float* B, float* C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * width + col;

    if (row < width && col < width) {
        C[idx] = A[idx] + B[idx];
    }
}

void cpuMatrixAdd(const float* A, const float* B, float* C, int width) {
    for (int i = 0; i < width * width; ++i) {
        C[i] = A[i] + B[i];
    }
}

void launchMatrixAddGPU(const float* A, const float* B, float* C, int width) {
    size_t bytes = width * width * sizeof(float);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((width + 15) / 16, (width + 15) / 16);

    matrixAddKernel<<<grid, block>>>(d_A, d_B, d_C, width);
    hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
